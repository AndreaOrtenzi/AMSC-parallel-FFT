#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdio>
#include "utils.cu"

__constant__ unsigned int bitReverseArr[BLOCK_SIZE] = {0, 4, 2, 6, 1, 5, 3, 7};

__device__ void fft8(hipFloatComplex &localData) {
    // Number of points
    constexpr unsigned int numBits = LOG_BLOCK_SIZE;

    // Each thread index
    const unsigned int tidBlockY = threadIdx.y * blockDim.x;
    const unsigned int tidX = threadIdx.x;

    /* 
    -----------------------------------------
        Step 1: Bit-Reversal Permutation
    -----------------------------------------
    */
    unsigned int j = bitReverseArr[tidX]; // Checked
    __syncwarp();

    // Use shuffle instead of writing back to data (data[j] = x)
    /*
        In shuffle operations each thread of the warp is represented as a lane.
        t0  ------------------------------
        t1  ------------------------------
        t2  ------------------------------
        ...
        t31 ------------------------------
        In this case j is from 0 to 7 because the 32 threads are divided into 4 rows of 8 elements.
        To correctly exchange the values of x and xj, we need to add 8*32//warpId to j.
        Notice that
        - 0-31 numbers in binary are 00000-11111
            Warp id can then be calculated taking the first 5 bits of the localThreadId
        - 0-7 numbers in binary are 000-111
            From this we can see that the first 2 bits identify the row and the last 3 bits identify the column
        We can then calculate the correct j by adding the 4th and 5th bits of the localThreadId to the j.
        Apply a mask 00..0011000 = 0x18 HEX = 24 DEC
    */
    const unsigned int maskedLocalIdx = (tidBlockY + tidX) & 0x18;
    j += maskedLocalIdx;

    // Use shuffle to get the value of thread j
    localData.x = __shfl_sync(0xFF, localData.x, j);
    localData.y = __shfl_sync(0xFF, localData.y, j);

    /*
    -----------------------------------------
        Step 2: Iterative FFT Stages
    -----------------------------------------
    */
    size_t maskRootsIdx = 0;
    for (unsigned s = 0; s < numBits; s++) { // Three stages for FFT of size 8

        // Synchronize threads before starting the stage
        __syncwarp();
        
        // Toggle the s-1 bit for the butterfly group
        unsigned pairedIdx = threadIdx.x ^ (1U << s);
        
        // This value is 1 if it has element u in the butterfly group, 0 otherwise
        hipFloatComplex hasElemU = make_hipFloatComplex(static_cast<float>(threadIdx.x < pairedIdx),0.);
        // This value is 1 if it has element t in the butterfly group, 0 otherwise
        hipFloatComplex hasElemT = make_hipFloatComplex(static_cast<float>(threadIdx.x > pairedIdx),0.);

        // TODO: Explain this
        // 8 - to reverse the order of the roots of unity
        // First root is duplicated in the 8th index
        // hipFloatComplex wm = rootsOfUnity[8 - (threadIdx.x % halfStep)*(8/step)];
        // hipFloatComplex wm = rootsOfUnity[(threadIdx.x % halfStep)*(BLOCK_SIZE/step)];// 0-7 % (1,2,4,8) * (8/2 = 4, 2=8/4, 1=8/8)
        
        // 000 001 010 011 100 101 110 111 -> 000
        // 000 001 010 011 100 101 110 111 %2 -> look at the last bit
        // 000 001 000 001 000 001 000 001 %4 -> look at the last 2 bits
        // 8 >> 1 = 4, 8 >> 2 = 2, 8 >> 3 = 1
        // *4 = << 2, *2 = << 1, *1 = << 0
        // numBits - s - 1 = 2, 1, 0        
        size_t rootsOfUnityIdx = threadIdx.x & maskRootsIdx; // threadIdx.x % halfStep
        rootsOfUnityIdx <<= numBits - s - 1; // *(BLOCK_SIZE/step)

        hipFloatComplex wm = rootsOfUnity[rootsOfUnityIdx];
        maskRootsIdx += 1<<s;

        hipFloatComplex pairedData;
        pairedData.x = __shfl_sync(0xFF, localData.x, pairedIdx + maskedLocalIdx);
        pairedData.y = __shfl_sync(0xFF, localData.y, pairedIdx + maskedLocalIdx);

        // t if hasElemU, -t if hasElemT
        hipFloatComplex t = hipCsubf(hipCmulf(hipCmulf(wm, pairedData), hasElemU), hipCmulf(hasElemT, hipCmulf(wm, localData)));
        // u if hasElemT, u if hasElemU
        hipFloatComplex u = hipCaddf(hipCmulf(localData, hasElemU), hipCmulf(pairedData, hasElemT));
        
        localData = hipCaddf(t, u);

        #if DEBUG
            if (tidBlockY == 0) {
                printf("Thread tidx= %d; tidBlocky = %d; s = %d; u = (%.2f,%.2f); t = (%.2f,%.2f)\n", tidX, tidBlockY, s, u.x, u.y, t.x, t.y);
            }
        #endif
    }
}


__global__ void fftQuantizeKernel(const hipFloatComplex *input, hipFloatComplex *output,const int width, const int height) {

    // Shared memory for an 8x8 subblock
    __shared__ hipFloatComplex blockData[BLOCK_SIZE][BLOCK_SIZE];

    // Load data into shared memory
    const unsigned int localX = threadIdx.x;
    const unsigned int localY = threadIdx.y;
    const unsigned int localIdx = localY * BLOCK_SIZE + localX;

    // Version with contiguous 8x8 submatrices    
    const unsigned int firstTileOffset = blockIdx.x * NUM_TILE_X_THREAD_BLOCK * BLOCK_SIZE * BLOCK_SIZE; // Offset for the first tile to process
    unsigned int globalIdx = firstTileOffset + localIdx;

    for (int i = 0; i < NUM_TILE_X_THREAD_BLOCK && globalIdx < width*height; ++i) {
        hipFloatComplex localData = input[globalIdx];

        // Perform FFT on rows
        fft8(localData);

        #if DEBUG
        __syncthreads();
        if (globalIdx==0) {
            for (int row = 0; row < BLOCK_SIZE; ++row) {
                for (int col = 0; col < BLOCK_SIZE; ++col) {
                    printf("(%.2f,%.2f) ", blockData[row][col].x, blockData[row][col].y);
                }
                printf("\n");
            }
        }
        __syncthreads();
        #endif

        // Transpose the block for column-wise FFT
        blockData[localX][localY] = localData;
        __syncthreads();
        localData = blockData[localY][localX];

        // Perform FFT on columns
        fft8(localData);

        // Transpose back
        blockData[localX][localY] = localData;
        __syncthreads();
        localData = blockData[localY][localX];
        
        // Apply quantization and write back to global memory
        output[globalIdx] = hipCdivf(localData, quantizationMatrix[localIdx]);
        globalIdx += BLOCK_SIZE * BLOCK_SIZE;
    }
}